#include "hip/hip_runtime.h"
#include "cracker.cuh"
#include "md5.cuh"

using namespace std;
extern __shared__ char s[];
typedef unsigned long long ull;
// const unsigned int m = 1 << 30, n = 62;
const unsigned int m = 128, n = 64;
// const unsigned int m = 32, n = 62;

__global__ void thread_attempt( char *secret, char *result, bool *found )
{
    const unsigned tid = threadIdx.x;

    // Load secret hash & charset into shared memory
    if (tid < word) s[tid] = secret[tid];
    if (tid < charset_sz) s[word + tid] = charset[tid];
    
    __syncthreads();
    
    // We check result before writing to avoid race condition
    unsigned count = 0;
    bool res;
    do
    {
        res = MD5(s, &count);
    } while (res == 0 && *found == 0);

    if (res)
    {
        // Used to signal to the remaining threads that the password has been found
        *found = true;
        // Copy the password to the result
        memcpy(result, s, word);
    }

}

#include "md5.h"

// Returns the password 
__host__ float crack( char *secret, char *deciphered )
{
    float delta;
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    char *data, *result;
    hipMalloc((void**)&data, sizeof(char) * MD5::size);
    hipMalloc((void**)&result, sizeof(char) * MD5::size);
    hipMemcpy(data, secret, sizeof(char) * MD5::size, hipMemcpyHostToDevice);

    bool *found;
    hipMalloc((void**) &found, sizeof(bool));

    // Start the timer
    hipEventRecord(start);
    
    // Run the cracking algorithm
    thread_attempt<<<m, n, MD5::hash_sz + charset_sz * sizeof(char) + sizeof(ull)>>>(data, result, found);
    hipDeviceSynchronize();

    // Stop the timer
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate the time delta
    hipEventElapsedTime(&delta, start, stop);

    bool status;
    hipMemcpy(&status, found, sizeof(bool), hipMemcpyDeviceToHost);
    hipFree(found);

    hipMemcpy(deciphered, result, sizeof(char) * MD5::size, hipMemcpyDeviceToHost);
    
    hipFree(data);
    hipFree(result);

    return delta;
}
